#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <stdint.h>
#include <time.h>
#include <string.h>
#include <omp.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <sys/types.h>
#include <sys/ipc.h>
#include <sys/shm.h>
#include <sys/time.h>
#include <unistd.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <complex.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <time.h>
#include <thread>

extern "C" {
    #include "read_file.h"
}

using namespace std;

#define _FILE_OFFSET_BITS 64
#define NX 2048
#define BATCH 1

void oops(int linenum, const char * msg) {
	fprintf(stderr, "\n*** OOPS, fatal error detected at line %d !!\n*** %s !!\n\n", linenum, msg);
	exit(86);
}

void run_cuFFT(char *arr, int NUM_IFFTS, int NUM_STREAMS, int gpu_id, fstream& out_file){

    //1D Batched IFFTS
    int rank = 1;                           // --- 1D FFTs
    int n[] = { 2*NX };                 // --- Size of the Fourier transform
    int istride = 1, ostride = 1;           // --- Distance between two successive input/output elements
    int idist = NX+1, odist = 2*NX; // --- Distance between batches for input and output respectively
    int inembed[] = { 0 };                  // --- Input size with pitch (ignored for 1D transforms)
    int onembed[] = { 0 };                  // --- Output size with pitch (ignored for 1D transforms)
    int batch = NUM_IFFTS;                  // Number of batches or IFFTS
    float *imag = new float[1];

    imag[0] = 0.f;

    char * buffer = arr;
    //Allocate input data
    printf("Allocating input data\n");
    hipfftComplex **h_in; 
    h_in = new hipfftComplex *[NUM_STREAMS];
    for (int ii = 0; ii < NUM_STREAMS; ii++) {
        h_in[ii] = new hipfftComplex [NUM_IFFTS*(NX+1)];
        for (int jj = 0; jj < NUM_IFFTS; jj++) {
            for (int kk = 0; kk < NX; kk++) {
                h_in[ii][jj*(NX+1) + kk].x = (float)buffer[ii*NUM_IFFTS*2*NX + jj*2*NX + 2*kk];
                h_in[ii][jj*(NX+1) +kk].y = (float)buffer[ii*NUM_IFFTS*2*NX + jj*2*NX + 2*kk + 1];
            }
            //For the last (2049th) complex number, set real part to imaginary part of 0th complex number, set imag to 0 and set imag of 0th complex number to 0
            h_in[ii][jj*(NX+1)+NX].x = h_in[ii][jj*(NX+1)].y;
            h_in[ii][jj*(NX+1)+NX].y = 0.f;
            h_in[ii][jj*(NX+1)].y = 0.f;
        }
    }
    printf("Allocated input data\n");

    //Allocate output data
    printf("Allocating output data\n");
    hipfftReal **h_out; 
    h_out = new hipfftReal *[NUM_STREAMS];
    for (int ii = 0; ii < NUM_STREAMS; ii++) {
        h_out[ii] = new hipfftReal [NUM_IFFTS*2*NX];
        for (int jj = 0; jj < NUM_IFFTS*2*NX; jj++) {
                h_out[ii][jj]= 0.f;
        }
    }
    printf("Allocated output data\n");

    // Pin host input and output memory for hipMemcpyAsync.
    printf("Pinning host input and output moemory for hipMemcpyAsync\n");
    for (int ii = 0; ii < NUM_STREAMS; ii++) {
            hipHostRegister(h_in[ii], NUM_IFFTS*(NX+1)*sizeof(hipfftComplex), hipHostRegisterPortable);
            hipHostRegister(h_out[ii], NUM_IFFTS*2*NX*sizeof(hipfftReal), hipHostRegisterPortable);

    }
    printf("Pinned host input and output moemory for hipMemcpyAsync\n");

    //Allocate pointers to device input and output arrays
    printf("Allocating pointers to device input and output arrays\n");
    hipfftComplex **d_in = new hipfftComplex *[NUM_STREAMS];
    hipfftReal **d_out = new hipfftReal *[NUM_STREAMS];
    printf("Allocated pointers to device input and output arrays\n");

    // Allocate intput and output arrays on device.
    printf("Allocating input and output arrays on device\n");
    for (int ii = 0; ii < NUM_STREAMS; ii++) {
        // d_in[ii] = new hipfftComplex [NUM_IFFTS*NX];
        // d_out[ii] = new hipfftComplex [NUM_IFFTS*NX];
        
        hipMalloc((void**)&d_in[ii], NUM_IFFTS*(NX+1)*sizeof(hipfftComplex));
        hipMalloc((void**)&d_out[ii], NUM_IFFTS*2*NX*sizeof(hipfftReal));

    }
    
    printf("Allocated input and output arrays on device\n");

    // Create CUDA streams.
    printf("Creating CUDA Streams\n");
    hipStream_t streams[NUM_STREAMS];
    for (int ii = 0; ii < NUM_STREAMS; ii++) {
        hipStreamCreate(&streams[ii]);
    }
    printf("Created CUDA streams\n");

    // Creates cuFFT plans and sets them in streams
    printf("Creating cuFFT plans and setting them in Streams\n");
    hipfftHandle* plans = (hipfftHandle*) malloc(sizeof(hipfftHandle)*NUM_STREAMS);
    for (int ii = 0; ii < NUM_STREAMS; ii++) {
            hipfftPlanMany(&plans[ii], rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2R, batch);
            hipfftSetStream(plans[ii], streams[ii]);
    }
    printf("Created cuFFT plans and setting them in Streams\n");

    // Fill streams with async memcopies and FFTs.
    printf("Filling Streams and RUNNING HIPFFT_BACKWARD\n");
    for (int ii = 0; ii < NUM_STREAMS; ii++) {

        hipMemcpyAsync(d_in[ii], h_in[ii], NUM_IFFTS*(NX+1)*sizeof(hipfftComplex), hipMemcpyHostToDevice, streams[ii]);
        hipfftExecC2R(plans[ii], (hipfftComplex*)d_in[ii], (hipfftReal*)d_out[ii]);
        hipMemcpyAsync(h_out[ii], d_out[ii], NUM_IFFTS*2*NX*sizeof(hipfftReal), hipMemcpyDeviceToHost, streams[ii]);
    }
    printf("Filled Streams AND RAN HIPFFT_BACKWARD \n");

    // Wait for calculations to complete.
    printf("Synchronising Streams\n");
    for(int ii = 0; ii < NUM_STREAMS; ii++) {
        hipStreamSynchronize(streams[ii]);
    }
    printf("Synchronised Streams\n");
    
    //Normalising Output (to be added)
    printf("Normalising Output\n");
    for (int ii = 0; ii < NUM_STREAMS; ii++) {
        for (int jj = 0; jj < NUM_IFFTS*2*NX; jj++) {
            h_out[ii][jj] = h_out[ii][jj]/(float)(2*NX);
        }
    }
    printf("Normalised Output\n");

    //Printing a few output values 
    // for (int ii = 0; ii < NUM_STREAMS; ii++) {
    //     printf("Printing for STREAM %d\n", ii);
    //     for (int jj = 0; jj < 3; jj++) {
    //         // printf("Printing for STR =  %d\n", jj);
    //         printf("OG : %f %f  Inverse %f %f \n",h_in[ii][jj].x, h_in[ii][jj].y,h_out[ii][jj], 0.f);
    //             // printf("%s %s \n", (char*) &h_out[ii][jj][kk].x, (char*) &h_out[ii][jj][kk].y);
            
    //     }
        
    // }

    //Writing to File
    printf("Seeking end of file\n");
    out_file.seekg(0, ios_base::end); //Seek end of file to start writing
    printf("Went to end of file\n");
    printf("********Writing to file********\n");
    for (int ii = 0; ii < NUM_STREAMS; ii++) {
        for (int jj = 0; jj < NUM_IFFTS*2*NX; jj++) {
                out_file.write((char *)&h_out[ii][jj], sizeof(char));
                out_file.write((char*) &imag[0], sizeof(char)); 
        }
    }
    printf("********Done Writing to file!********\n");

    // Free memory and streams.
    printf("Freeing memory and streams\n");
    for (int ii = 0; ii < NUM_STREAMS; ii++) {
        hipHostUnregister(h_in[ii]);
        hipHostUnregister(h_out[ii]);
        hipFree(d_in[ii]);
        hipFree(d_out[ii]);
        delete[] h_in[ii];
        delete[] h_out[ii];
        
        hipStreamDestroy(streams[ii]);
    }
    printf("Freed memory and streams\n");

    delete plans;

    hipDeviceReset(); 

    
}

int main(int argc, const char *argv[])
{
    time_t start, stop;
    // char *infile, *buffer, *buffer2, *buffer3, *buffer4; 
	// short int *data1_int;
	// int op_file;
	// const double time_per_block=671088.64; // uSec.
    const int NUM_IFFTS = 4069;
    
    const int NUM_STREAMS = 3;

    //NUM_IFFTS*NUM_STREAMS = num_blocks

	if(argc != 4) {
		printf("Invalid number of parameters! \n");
		printf("Usage: ./%s  <input file> <output file> <GPU ID>\n",argv[0]);
		exit(-1);
	}

    int gpu_id = atoi(argv[3]);
    if(hipSetDevice(gpu_id) != hipSuccess)
        oops(__LINE__, "hipSetDevice FAILED\n");

    // int iterations = atoi(argv[2]);
    
    




    
    // infile = read_file(argv[1]);
    
    fstream out_file;
    out_file.open(argv[2], ios::binary | ios::out);
    if(!out_file)
   {
       cout<<"Error in creating file!!!";
       return 0;
   }
  
   cout<<"File created successfully.\n";

    // buffer = infile;
    // buffer2 = buffer + 30510*2048; 
    // buffer3 = buffer2 + 30510*2048;
    // buffer4 = buffer3 + 30510*2048; 

    start = time(NULL);
    //Read file in chunks
    size_t block_size = 4096;
    size_t num_blocks = 12207;
    int file_size;
    double total_num_blocks;

    // char* file_contents = new char [block_size*num_blocks];

    //Open Input File
    ifstream in_file;
    in_file.open(argv[1], ios::binary| ios::in);

    //Get File Size
    in_file.seekg(0,ios::end);
    file_size = in_file.tellg();
    in_file.seekg(0, ios::beg);

    total_num_blocks = file_size/(double)(4096);

    printf("The total number of blocks is %d\n", (int) total_num_blocks);

    //Read data into buffer
    for (int i = 0; i < 10; i++) {
        char* file_contents = new char [block_size*num_blocks];
        in_file.read(file_contents, block_size*num_blocks);

        printf("Running for buffer %d\n", i);
        
        run_cuFFT(file_contents, NUM_IFFTS, NUM_STREAMS, gpu_id, out_file);
        delete[] file_contents;
        printf("Done for buffer %d\n",i);
    }
    out_file.close();
    in_file.close();
    stop = time(NULL);
    printf("The number of seconds for to run was %ld\n", stop - start);
    printf("The total number of blocks is %d\n", (int) total_num_blocks);

    return 0;
}