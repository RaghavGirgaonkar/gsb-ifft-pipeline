#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <stdint.h>
#include <time.h>
#include <string.h>
#include <omp.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <sys/types.h>
#include <sys/ipc.h>
#include <sys/shm.h>
#include <sys/time.h>
#include <unistd.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <complex.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <time.h>
#include <thread>

extern "C" {
    #include "read_file.h"
}

using namespace std;

#define _FILE_OFFSET_BITS 64
#define NX 2048
#define BATCH 1


void oops(int linenum, const char * msg) {
	fprintf(stderr, "\n*** OOPS, fatal error detected at line %d !!\n*** %s !!\n\n", linenum, msg);
	exit(86);
}

void run_cuFFT(char *arr, int NUM_IFFTS, int NUM_STREAMS, int gpu_id, fstream& out_file){

    char * buffer = arr;
    //Allocate input data
    printf("Allocating input data\n");
    hipfftComplex ***h_in; 
    h_in = new hipfftComplex **[NUM_STREAMS];
    for (int ii = 0; ii < NUM_STREAMS; ii++) {
        h_in[ii] = new hipfftComplex *[NUM_IFFTS];
        for (int jj = 0; jj < NUM_IFFTS; jj++) {
            h_in[ii][jj] = new hipfftComplex[NX];
            for (int kk = 0; kk < NX; kk++) {
                h_in[ii][jj][kk].x = (float)*buffer++;
                h_in[ii][jj][kk].y = (float)*buffer++;
            }
        }
    }
    printf("Allocated input data\n");

    //Allocate output data
    printf("Allocating output data\n");
    hipfftComplex ***h_out; 
    h_out = new hipfftComplex **[NUM_STREAMS];
    for (int ii = 0; ii < NUM_STREAMS; ii++) {
        h_out[ii] = new hipfftComplex *[NUM_IFFTS];
        for (int jj = 0; jj < NUM_IFFTS; jj++) {
            h_out[ii][jj] = new hipfftComplex[NX];
            for (int kk = 0; kk < NX; kk++) {
                h_out[ii][jj][kk].x = 0.f;
                h_out[ii][jj][kk].y = 0.f;
            }
        }
    }
    printf("Allocated output data\n");

    // Pin host input and output memory for hipMemcpyAsync.
    printf("Pinning host input and output moemory for hipMemcpyAsync\n");
    for (int ii = 0; ii < NUM_STREAMS; ii++) {
        for (int jj = 0; jj < NUM_IFFTS; jj++) {
            hipHostRegister(h_in[ii][jj], NX*sizeof(hipfftComplex), hipHostRegisterPortable);
            hipHostRegister(h_out[ii][jj], NX*sizeof(hipfftComplex), hipHostRegisterPortable);

        }
    }
    printf("Pinned host input and output moemory for hipMemcpyAsync\n");

    //Allocate pointers to device input and output arrays
    printf("Allocating pointers to device input and output arrays\n");
    hipfftComplex ***d_in = new hipfftComplex **[NUM_STREAMS];
    hipfftComplex ***d_out = new hipfftComplex **[NUM_STREAMS];
    printf("Allocated pointers to device input and output arrays\n");

    // Allocate intput and output arrays on device.
    printf("Allocating input and output arrays on device\n");
    for (int ii = 0; ii < NUM_STREAMS; ii++) {
        d_in[ii] = new hipfftComplex *[NUM_IFFTS];
        d_out[ii] = new hipfftComplex *[NUM_IFFTS];
        for (int jj = 0; jj < NUM_IFFTS; jj++) {
            hipMalloc((void**)&d_in[ii][jj], NX*sizeof(hipfftComplex));
            hipMalloc((void**)&d_out[ii][jj], NX*sizeof(hipfftComplex));
        }
    }
    
    printf("Allocated input and output arrays on device\n");

    // Create CUDA streams.
    printf("Creating CUDA Streams\n");
    hipStream_t streams[NUM_STREAMS];
    for (int ii = 0; ii < NUM_STREAMS; ii++) {
        hipStreamCreate(&streams[ii]);
    }
    printf("Created CUDA streams\n");

    // Creates cuFFT plans and sets them in streams
    printf("Creating cuFFT plans and setting them in Streams\n");
    hipfftHandle* plans = (hipfftHandle*) malloc(sizeof(hipfftHandle)*NUM_STREAMS*NUM_IFFTS);
    for (int ii = 0; ii < NUM_STREAMS; ii++) {
        for (int jj = 0; jj < NUM_IFFTS; jj++) {
            hipfftPlan1d(&plans[ii*NUM_IFFTS + jj], NX, HIPFFT_C2C, 1);
            hipfftSetStream(plans[ii*NUM_IFFTS + jj], streams[ii]);
        }
    }
    printf("Created cuFFT plans and setting them in Streams\n");

    // Fill streams with async memcopies and FFTs.
    printf("Filling Streams and RUNNING HIPFFT_BACKWARD\n");
    for (int ii = 0; ii < NUM_STREAMS; ii++) {
        for (int jj = 0; jj < NUM_IFFTS; jj++) {

            hipMemcpyAsync(d_in[ii][jj], h_in[ii][jj], NX*sizeof(hipfftComplex), hipMemcpyHostToDevice, streams[ii]);
            hipfftExecC2C(plans[ii*NUM_IFFTS + jj], (hipfftComplex*)d_in[ii][jj], (hipfftComplex*)d_out[ii][jj], HIPFFT_BACKWARD);
            hipMemcpyAsync(h_out[ii][jj], d_out[ii][jj], NX*sizeof(hipfftComplex), hipMemcpyDeviceToHost, streams[ii]);
        }
    }
    printf("Filled Streams AND RAN HIPFFT_BACKWARD \n");

    // Wait for calculations to complete.
    printf("Synchronising Streams\n");
    for(int ii = 0; ii < NUM_STREAMS; ii++) {
        hipStreamSynchronize(streams[ii]);
    }
    printf("Synchronised Streams\n");
    
    //Normalising Output (to be added)
    printf("Normalising Output\n");
    for (int ii = 0; ii < NUM_STREAMS; ii++) {
        for (int jj = 0; jj < NUM_IFFTS; jj++) {
            for (int kk = 0; kk < NX; kk++) {
                h_out[ii][jj][kk].x = h_out[ii][jj][kk].x/(float)NX;
                h_out[ii][jj][kk].y = h_out[ii][jj][kk].y/(float)NX;
            }
        }
    }
    printf("Normalised Output\n");

    //Printing a few output values 
    // for (int ii = 0; ii < NUM_STREAMS; ii++) {
    //     printf("Printing for STREAM %d\n", ii);
    //     for (int jj = 0; jj < NUM_IFFTS; ++jj) {
    //         printf("Printing for NUM_IFFT =  %d\n", jj);
    //         for (int kk = 0; kk < 3; ++kk) {
    //             // printf("OG : %f %f  Inverse %f %f \n",h_in[ii][jj][kk].x, h_in[ii][jj][kk].y,h_out[ii][jj][kk].x,h_out[ii][jj][kk].y);
    //             printf("%s %s \n", (char*) &h_out[ii][jj][kk].x, (char*) &h_out[ii][jj][kk].y);
    //         }
    //     }
    // }

    //Writing to File
    printf("Seeking end of file\n");
    out_file.seekg(0, ios_base::end); //Seek end of file to start writing
    printf("Went to end of file\n");
    printf("********Writing to file********\n");
    for (int ii = 0; ii < NUM_STREAMS; ii++) {
        for (int jj = 0; jj < NUM_IFFTS; jj++) {
            for (int kk = 0; kk < NX; kk++) {
                out_file.write((char*)&h_out[ii][jj][kk].x, sizeof(char));
                out_file.write((char*)&h_out[ii][jj][kk].y, sizeof(char)); 
            }
        }
    }
    printf("********Done Writing to file!********\n");

    // Free memory and streams.
    printf("Freeing memory and streams\n");
    for (int ii = 0; ii < NUM_STREAMS; ii++) {
        for (int jj = 0; jj < NUM_IFFTS; jj++) {
        hipHostUnregister(h_in[ii][jj]);
        hipHostUnregister(h_out[ii][jj]);
        hipFree(d_in[ii][jj]);
        hipFree(d_out[ii][jj]);
        delete[] h_in[ii][jj];
        delete[] h_out[ii][jj];
        
        }
        hipStreamDestroy(streams[ii]);
    }
    printf("Freed memory and streams\n");

    delete plans;

    hipDeviceReset(); 

    
}

int main(int argc, const char *argv[])
{
    time_t start, stop;
    char *infile, *buffer, *buffer2, *buffer3, *buffer4; 
	// short int *data1_int;
	// int op_file;
	// const double time_per_block=671088.64; // uSec.
	int nint;
	int data_size;
    const int NUM_IFFTS = 10;
    const int NUM_STREAMS = 339;

	if(argc != 4) {
		printf("Invalid number of parameters! \n");
		printf("Usage: ./%s  <input file> <output file> <GPU ID>\n",argv[0]);
		exit(-1);
	}

    int gpu_id = atoi(argv[3]);
    if(hipSetDevice(gpu_id) != hipSuccess)
        oops(__LINE__, "hipSetDevice FAILED\n");

    // int iterations = atoi(argv[2]);

	nint= 3;
	data_size = 2*2048*nint; // 2 for real & Imaginary data.
 
    
    infile = read_file(argv[1]);
    
    fstream out_file;
    out_file.open(argv[2], ios::binary | ios::out);
    if(!out_file)
   {
       cout<<"Error in creating file!!!";
       return 0;
   }
  
   cout<<"File created successfully.\n";

    buffer = infile;
    buffer2 = buffer + 30510*2048; 
    buffer3 = buffer2 + 30510*2048;
    buffer4 = buffer3 + 30510*2048; 

    start = time(NULL);

    

    for(int i = 0; i < 36; i++){
       run_cuFFT(buffer, NUM_IFFTS, NUM_STREAMS, gpu_id, out_file);
       buffer += NUM_IFFTS*NUM_STREAMS*NX;
       printf("Ran for buffer%d\n", i+1); 
    }

    out_file.close();

    stop = time(NULL);
    printf("The number of seconds for to run was %ld\n", stop - start);

    return 0;
}