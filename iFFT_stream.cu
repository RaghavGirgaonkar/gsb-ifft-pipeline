#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <stdint.h>
#include <time.h>
#include <string.h>
#include <omp.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <sys/types.h>
#include <sys/ipc.h>
#include <sys/shm.h>
#include <sys/time.h>
#include <unistd.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <complex.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <time.h>
#include <thread>

extern "C" {
    #include "read_file.h"
}

using namespace std;

#define _FILE_OFFSET_BITS 64
#define NX 2048
#define BATCH 1


void oops(int linenum, const char * msg) {
	fprintf(stderr, "\n*** OOPS, fatal error detected at line %d !!\n*** %s !!\n\n", linenum, msg);
	exit(86);
}

void run_test(char *arr, int num_iter, int deviceID){
    printf("\n------------------------------------------------ run_test: Starting on GPU ID \t %d\n",deviceID);
    if(hipSetDevice(deviceID) != hipSuccess)
        oops(__LINE__, "hipSetDevice FAILED\n");
    char * arrptr = arr;
    // Allocate host memory for the signal
    int malloc_size = sizeof(hipfftComplex) * NX;
    for(unsigned int j = 0; j< num_iter; j++){
    printf("Using malloc size for h_signal%d\n", malloc_size);
    hipfftComplex* h_signal = (hipfftComplex*)malloc(malloc_size);
    if (h_signal == NULL)
        oops(__LINE__, "malloc 1 FAILED\n");
    // hipfftComplex* h_signal2 = (hipfftComplex*)malloc(malloc_size);
    // if (h_signal2 == NULL)
    //     oops(__LINE__, "malloc 2 FAILED\n");

    // Initalize the memory for the signal
    
    // char * arrptr2 = arr + 4096;
    for (unsigned int i = 0; i < NX; i++) {
        h_signal[i].x = (float)*arrptr++;
        h_signal[i].y = (float)*arrptr++;
        // h_signal2[i].x = (float)*arrptr2++;
        // h_signal2[i].y = (float)*arrptr2++;
    }
    printf("Done populating h_signals\n");
    
    // Allocate device memory for signal
    size_t dev_mem_size = sizeof(hipfftComplex) * NX;
    printf("Device memory allocation size for each signal = %ld\n", dev_mem_size);
    hipfftComplex* d_signal;
    // hipfftComplex* d_signal2;
    if(hipMalloc((void**)&d_signal, dev_mem_size) != hipSuccess)
       oops(__LINE__, "hipMalloc d_signal FAILED\n");
    printf("d_signal allocated\n");
    // if(hipMalloc((void**)&d_signal2, dev_mem_size) != hipSuccess)
    //    oops(__LINE__, "hipMalloc d_signal2 FAILED\n");
    // printf("d_signal2 allocated\n");

    // Copy host memory to device
    if(hipMemcpy(d_signal, h_signal, dev_mem_size, hipMemcpyHostToDevice) != hipSuccess)
       oops(__LINE__, "hipMemcpy to d_signal FAILED\n");
    printf("Done copying from Host to Device pt 1\n");
    // if(hipMemcpy(d_signal2, h_signal2, dev_mem_size, hipMemcpyHostToDevice) != hipSuccess)
    //    oops(__LINE__, "hipMemcpy to d_signal2 FAILED\n");
    // printf("Done copying from Host to Device pt 2\n");

    //Create Plan
    hipfftHandle plan;
    if(hipfftPlan1d(&plan, NX, HIPFFT_C2C, BATCH) != HIPFFT_SUCCESS)
        oops(__LINE__, "hipfftPlan1d FAILED\n");
   
    printf("Made plan\n");

    //Execute Inverse Transform
    if (hipfftExecC2C(plan, (hipfftComplex *)d_signal, (hipfftComplex *)d_signal, HIPFFT_BACKWARD) != HIPFFT_SUCCESS)
	    oops(__LINE__, "hipfftExecC2C d_signal FAILED");
    // if (hipfftExecC2C(plan, (hipfftComplex *)d_signal2, (hipfftComplex *)d_signal2, HIPFFT_BACKWARD) != HIPFFT_SUCCESS)
	//     oops(__LINE__, "hipfftExecC2C d_signal2 FAILED");
    printf("Done with iFFT\n");

    // Synchronize device.
    if(hipDeviceSynchronize() != hipSuccess)
	    oops(__LINE__, "hipDeviceSynchronize FAILED");
    
    //Copy device memory to Host
    hipfftComplex* h_inverse_signal = (hipfftComplex*)malloc(sizeof(hipfftComplex) * NX);
    if(h_inverse_signal == NULL)
        oops(__LINE__, "malloc h_inverse_signal FAILED");
    // hipfftComplex* h_inverse_signal2 = (hipfftComplex*)malloc(sizeof(hipfftComplex) * NX);
    // if(h_inverse_signal2 == NULL)
    //     oops(__LINE__, "malloc h_inverse_signal2 FAILED");
    if(hipMemcpy(h_inverse_signal, d_signal, dev_mem_size,hipMemcpyDeviceToHost) != hipSuccess)
        oops(__LINE__, "hipMemcpy h_inverse_signal FAILED");
    // if(hipMemcpy(h_inverse_signal2, d_signal2, dev_mem_size,hipMemcpyDeviceToHost) != hipSuccess)
    //     oops(__LINE__, "hipMemcpy h_inverse_signal2 FAILED");
    printf("Done with copying back to Host\n");

    //Display Inverse Transform
    printf("Printing iFFT for first array:\n");
    for(int i=0;i< 3;i++){
        h_inverse_signal[i].x= h_inverse_signal[i].x/(float)NX;
        h_inverse_signal[i].y= h_inverse_signal[i].y/(float)NX;
        printf("OG : %f %f  Inverse %f %f \n",h_signal[i].x,h_signal[i].y,h_inverse_signal[i].x,h_inverse_signal[i].y);
    }
    // printf("Printing iFFT for second array:\n");
    // for(int i = 0; i < 3; i++){
    //     h_inverse_signal2[i].x= h_inverse_signal2[i].x/(float)NX;
    //     h_inverse_signal2[i].y= h_inverse_signal2[i].y/(float)NX;
    //     printf("OG : %f %f  Inverse %f %f \n",h_signal2[i].x,h_signal2[i].y,h_inverse_signal2[i].x,h_inverse_signal2[i].y);
    // }
    hipfftDestroy(plan);
    printf("Done destroying plan\n");
    free(h_signal);
    // free(h_signal2);
    printf("Done freeing hsignals\n");
    free(h_inverse_signal);
    // free(h_inverse_signal2);
    printf("Done freeing h_inverse signals\n");
    hipFree(d_signal);
    // hipFree(d_signal2);
    printf("Done freeing d_signals\n");
    hipDeviceReset();
    printf("Done hipDeviceReset()\n");
    }

}

int main(int argc, const char *argv[])
{
    time_t start, stop;
    char *infile, *buffer, *buffer2, *buffer3; 
	// short int *data1_int;
	// int op_file;
	// const double time_per_block=671088.64; // uSec.
	int nint;
	int data_size;
    const int NUM_IFFTS = 10;
    const int NUM_STREAMS = 5120;

	if(argc != 4) {
		printf("Invalid number of parameters! \n");
		printf("Usage: ./%s  <vlt file1> <num_iterations> <GPU ID>\n",argv[0]);
		exit(-1);
	}

    int gpu_id = atoi(argv[3]);
    if(hipSetDevice(gpu_id) != hipSuccess)
        oops(__LINE__, "hipSetDevice FAILED\n");

    int iterations = atoi(argv[2]);

	nint= 3;
	data_size = 2*2048*nint; // 2 for real & Imaginary data.
 
    
    infile = read_file(argv[1]);
    buffer = infile;
    buffer2 = buffer + 2*2048; 
    buffer3 = buffer2 + 2*2048; 

    start = time(NULL);

    //Allocate input data
    printf("Allocating input data\n");
    hipfftComplex **h_in = new hipfftComplex *[NUM_STREAMS];
    for (int ii = 0; ii < NUM_STREAMS; ii++) {
        h_in[ii] = new hipfftComplex[NX];
        for (int jj = 0; jj < NUM_IFFTS; ++jj) {
            h_in[ii][jj].x = (float)*buffer++;
            h_in[ii][jj].y = (float)*buffer++;
        }
    }
    printf("Allocated input data\n");

    //Allocate output data
    printf("Allocating output data\n");
    hipfftComplex **h_out = new hipfftComplex *[NUM_STREAMS];
    for (int ii = 0; ii < NUM_STREAMS; ii++) {
        h_out[ii] = new hipfftComplex[NX];
        for (int jj = 0; jj < NX; ++jj) {
            h_out[ii][jj].x = 0.f;
            h_out[ii][jj].y = 0.f;
        }
    }
    printf("Allocated output data\n");

    // Pin host input and output memory for hipMemcpyAsync.
    printf("Pinning host input and output moemory for hipMemcpyAsync\n");
    for (int ii = 0; ii < NUM_STREAMS; ii++) {
        hipHostRegister(h_in[ii], NX*sizeof(hipfftComplex), hipHostRegisterPortable);
        hipHostRegister(h_out[ii], NX*sizeof(hipfftComplex), hipHostRegisterPortable);
    }
    printf("Pinned host input and output moemory for hipMemcpyAsync\n");

    //Allocate pointers to device input and output arrays
    printf("Allocating pointers to device input and output arrays\n");
    hipfftComplex **d_in = new hipfftComplex *[NUM_STREAMS];
    hipfftComplex **d_out = new hipfftComplex *[NUM_STREAMS];
    printf("Allocated pointers to device input and output arrays\n");

    // Allocate intput and output arrays on device.
    printf("Allocating input and output arrays on device\n");
    for (int ii = 0; ii < NUM_STREAMS; ii++) {
        hipMalloc((void**)&d_in[ii], NX*sizeof(hipfftComplex));
        hipMalloc((void**)&d_out[ii], NX*sizeof(hipfftComplex));
    }
    printf("Allocated input and output arrays on device\n");

    // Create CUDA streams.
    printf("Creating CUDA Streams\n");
    hipStream_t streams[NUM_STREAMS];
    for (int ii = 0; ii < NUM_STREAMS; ii++) {
        hipStreamCreate(&streams[ii]);
    }
    printf("Created CUDA streams\n");

    // Creates cuFFT plans and sets them in streams
    printf("Creating cuFFT plans and setting them in Streams\n");
    hipfftHandle* plans = (hipfftHandle*) malloc(sizeof(hipfftHandle)*NUM_STREAMS);
    for (int ii = 0; ii < NUM_STREAMS; ii++) {
        hipfftPlan1d(&plans[ii], NX, HIPFFT_C2C, 1);
        hipfftSetStream(plans[ii], streams[ii]);
    }
    printf("Created cuFFT plans and setting them in Streams\n");

    // Fill streams with async memcopies and FFTs.
    printf("Filling Streams\n");
    for (int ii = 0; ii < NUM_STREAMS; ii++) {
        int jj = ii;
        hipMemcpyAsync(d_in[jj], h_in[jj], NX*sizeof(hipfftComplex), hipMemcpyHostToDevice, streams[jj]);
        hipfftExecC2C(plans[jj], (hipfftComplex*)d_in[jj], (hipfftComplex*)d_out[jj], HIPFFT_BACKWARD);
        hipMemcpyAsync(h_out[jj], d_out[jj], NX*sizeof(hipfftComplex), hipMemcpyDeviceToHost, streams[jj]);
    }
    printf("Filled Streams\n");

    // Wait for calculations to complete.
    printf("Synchronising Streams\n");
    for(int ii = 0; ii < NUM_STREAMS; ii++) {
        hipStreamSynchronize(streams[ii]);
    }
    printf("Synchronised Streams\n");
    
    //Normalising Output (to be added)

    //Printing a few output values 
    for (int ii = 0; ii < NUM_STREAMS; ii++) {
        printf("Printing for STREAM %d\n", ii);
        for (int jj = 0; jj < 3; ++jj) {
            printf("OG : %f %f  Inverse %f %f \n",h_in[ii][jj].x, h_in[ii][jj].y,h_out[ii][jj].x,h_out[ii][jj].y);
        }
    }

    // Free memory and streams.
    printf("Freeing memory and streams\n");
    for (int ii = 0; ii < NUM_STREAMS; ii++) {
        hipHostUnregister(h_in[ii]);
        hipHostUnregister(h_out[ii]);
        hipFree(d_in[ii]);
        hipFree(d_out[ii]);
        delete[] h_in[ii];
        delete[] h_out[ii];
        hipStreamDestroy(streams[ii]);
    }
    printf("Freed memory and streams\n");

    delete plans;

    hipDeviceReset(); 


    stop = time(NULL);
    printf("The number of seconds for to run was %ld\n", stop - start);

    return 0;
}